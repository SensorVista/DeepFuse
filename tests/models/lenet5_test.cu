#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <dnn/models/lenet5.cuh>

#include <vector>
#include <random>
#include <fstream>
#include <cstdio>
#include <filesystem>

namespace dnn {
namespace test {

class LeNet5Test : public ::testing::Test {
protected:
    void SetUp() override {
        // Set up any common test resources
    }

    void TearDown() override {
        // Clean up any common test resources
    }

    // Helper function to generate random input data
    std::vector<float> generate_random_input(size_t size) {
        std::vector<float> data(size);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(0.0f, 1.0f);
        
        for (size_t i = 0; i < size; ++i) {
            data[i] = dis(gen);
        }
        return data;
    }

    // Helper function to generate random target data
    std::vector<float> generate_random_target(size_t size) {
        std::vector<float> data(size);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<int> dis(0, 9);
        
        for (size_t i = 0; i < size; ++i) {
            data[i] = static_cast<float>(dis(gen));
        }
        return data;
    }
};

TEST_F(LeNet5Test, Constructor) {
    LeNet5<float> model;
    
    // Verify model architecture
    auto layers = model.parameters();
    EXPECT_GT(layers.size(), 0);
}

TEST_F(LeNet5Test, ForwardPass) {
    LeNet5<float> model;
    
    // Create input tensor (batch_size=1, channels=1, height=32, width=32)
    std::vector<int> input_shape = {1, 1, 32, 32};
    tensor<float> input(input_shape);
    
    // Fill input with random data
    std::vector<float> input_data = generate_random_input(static_cast<size_t>(input_shape[0] * input_shape[1] * input_shape[2] * input_shape[3]));
    input.upload(input_data.data());
    
    // Forward pass
    tensor<float> output = model.forward(input);
    
    // Verify output shape (should be [1, 10] for 10 classes)
    EXPECT_EQ(output.shape()[0], 1);
    EXPECT_EQ(output.shape()[1], 10);
    
    // Download and verify output values
    std::vector<float> output_data(output.size());
    output.download(output_data.data());
    
    // Check if output values are reasonable (not NaN or Inf)
    for (float val : output_data) {
        EXPECT_FALSE(std::isnan(val));
        EXPECT_FALSE(std::isinf(val));
    }
}

TEST_F(LeNet5Test, TrainingStep) {
    LeNet5<float> model(0.01, 0.9, true);
    
    // Create input tensor
    std::vector<int> input_shape = {1, 1, 32, 32};
    tensor<float> input(input_shape);
    
    // Create target tensor
    std::vector<int> target_shape = {1, 10};
    tensor<float> target(target_shape);
    
    // Fill input with random data
    std::vector<float> input_data = generate_random_input(static_cast<size_t>(input_shape[0] * input_shape[1] * input_shape[2] * input_shape[3]));
    input.upload(input_data.data());
    
    // Fill target with random one-hot encoded data
    std::vector<float> target_data = generate_random_target(static_cast<size_t>(target_shape[1]));
    target.upload(target_data.data());
    
    // Record initial loss
    float initial_loss = model.loss();
    
    // Perform training step
    model.train_step(input, target);
    
    // Record new loss
    float new_loss = model.loss();
    
    // Verify that loss has changed
    EXPECT_NE(initial_loss, new_loss);
}

TEST_F(LeNet5Test, MultipleTrainingSteps) {
    LeNet5<float> model(0.01, 0.9, true);
    
    // Create input tensor
    std::vector<int> input_shape = {1, 1, 32, 32};
    tensor<float> input(input_shape);
    
    // Create target tensor
    std::vector<int> target_shape = {1, 10};
    tensor<float> target(target_shape);
    
    // Perform multiple training steps
    std::vector<float> losses;
    for (int i = 0; i < 5; ++i) {
        // Fill input with random data
        std::vector<float> input_data = generate_random_input(static_cast<size_t>(input_shape[0] * input_shape[1] * input_shape[2] * input_shape[3]));
        input.upload(input_data.data());
        
        // Fill target with random one-hot encoded data
        std::vector<float> target_data = generate_random_target(static_cast<size_t>(target_shape[1]));
        target.upload(target_data.data());
        
        // Perform training step
        model.train_step(input, target);
        
        // Record loss
        losses.push_back(model.loss());
    }
    
    // Verify that loss is generally decreasing
    bool loss_decreased = false;
    for (size_t i = 1; i < losses.size(); ++i) {
        if (losses[i] < losses[i-1]) {
            loss_decreased = true;
            break;
        }
    }
    EXPECT_TRUE(loss_decreased);
}

TEST_F(LeNet5Test, SaveLoadRoundtrip) {
    LeNet5<float> model(0.01, 0.9, true);
    std::vector<int> input_shape = {1, 1, 32, 32};
    tensor<float> input(input_shape);
    std::vector<float> input_data = generate_random_input(static_cast<size_t>(input.size()));
    input.upload(input_data.data());
    std::vector<int> target_shape = {1, 10};
    tensor<float> target(target_shape);
    std::vector<float> target_data = generate_random_target(static_cast<size_t>(target.size()));
    target.upload(target_data.data());
    // Mutate model parameters by training
    model.train_step(input, target);
    // Save to a temp file
    std::string path = "lenet5_test_model.bin";
    model.save(path);
    // Load from file
    auto loaded = LeNet5<float>::load(path, true);
    // Compare parameters
    auto params1 = model.parameters();
    auto params2 = loaded->parameters();
    ASSERT_EQ(params1.size(), params2.size());
    for (size_t i = 0; i < params1.size(); ++i) {
        // Use approx_equal for floating point tolerance
        EXPECT_TRUE(params1[i]->approx_equal(*params2[i], 1e-5f)) << "Parameter " << i << " mismatch after load/save.";
    }
    // Clean up
    std::filesystem::remove(path);
}

TEST_F(LeNet5Test, LoadCorruptFileThrows) {
    std::string bad_path = "nonexistent_file.bin";
    EXPECT_THROW({
        auto m = LeNet5<float>::load(bad_path, true);
    }, std::runtime_error);
}

} // namespace test
} // namespace dnn 