#include "hip/hip_runtime.h"
#include "pooling_layer.cuh"
#include "../utils/common.cuh"
#include "../core/device.cuh"

#include <hip/hip_runtime.h>

namespace dnn {

#ifndef ENABLE_CUDNN
template<typename T>
__global__ void avg_pool_forward_2d(
    T* output, const T* input,
    int batch_size, int channels,
    int height, int width,
    int kernel_size, int stride,
    int out_h, int out_w
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * out_h * out_w) return;

    int b = idx / (channels * out_h * out_w);
    int c = (idx / (out_h * out_w)) % channels;
    int oh = (idx / out_w) % out_h;
    int ow = idx % out_w;

    T sum = 0.0f;
    int count = 0;

    for (int kh = 0; kh < kernel_size; ++kh) {
        for (int kw = 0; kw < kernel_size; ++kw) {
            int ih = oh * stride + kh;
            int iw = ow * stride + kw;
            if (ih < height && iw < width) {
                int in_idx = (
                    b * channels * height * width +
                    c * height * width +
                    ih * width + iw
                );
                sum += input[in_idx];
                ++count;
            }
        }
    }

    output[idx] = sum / static_cast<T>(count);
}

template<typename T>
__global__ void max_pool_forward_2d(
    T* output, const T* input,
    int batch_size, int channels,
    int height, int width,
    int kernel_size, int stride,
    int out_h, int out_w
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * out_h * out_w) return;

    int b = idx / (channels * out_h * out_w);
    int c = (idx / (out_h * out_w)) % channels;
    int oh = (idx / out_w) % out_h;
    int ow = idx % out_w;

    T max_val = utils::neg_infinity<T>();

    for (int kh = 0; kh < kernel_size; ++kh) {
        for (int kw = 0; kw < kernel_size; ++kw) {
            int ih = oh * stride + kh;
            int iw = ow * stride + kw;
            if (ih < height && iw < width) {
                int in_idx = (
                    b * channels * height * width +
                    c * height * width +
                    ih * width + iw
                );
                max_val = max(max_val, input[in_idx]);
            }
        }
    }

    output[idx] = max_val;
}

template<typename T>
__global__ void avg_pool_backward_2d(
    T* grad_input, const T* grad_output,
    int batch_size, int channels,
    int height, int width,
    int kernel_size, int stride,
    int out_h, int out_w
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * height * width) return;

    int b = idx / (channels * height * width);
    int c = (idx / (height * width)) % channels;
    int h = (idx / width) % height;
    int w = idx % width;

    int oh = h / stride;
    int ow = w / stride;

    if (oh < out_h && ow < out_w) {
        int out_idx = (
            b * channels * out_h * out_w +
            c * out_h * out_w +
            oh * out_w + ow
        );
        T grad = grad_output[out_idx] / static_cast<T>(kernel_size * kernel_size);
        atomicAdd(&grad_input[idx], grad);
    }
}

template<typename T>
__global__ void max_pool_backward_2d(
    T* grad_input, const T* grad_output, const T* input,
    int batch_size, int channels,
    int height, int width,
    int kernel_size, int stride,
    int out_h, int out_w
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size * channels * out_h * out_w) return;

    int b = idx / (channels * out_h * out_w);
    int c = (idx / (out_h * out_w)) % channels;
    int oh = (idx / out_w) % out_h;
    int ow = idx % out_w;

    int h_start = oh * stride;
    int w_start = ow * stride;

    T max_val = utils::neg_infinity<T>();
    int max_idx = -1;

    for (int kh = 0; kh < kernel_size; ++kh) {
        for (int kw = 0; kw < kernel_size; ++kw) {
            int ih = h_start + kh;
            int iw = w_start + kw;
            if (ih < height && iw < width) {
                int in_idx = (
                    b * channels * height * width +
                    c * height * width +
                    ih * width + iw
                    );
                if (input[in_idx] > max_val) {
                    max_val = input[in_idx];
                    max_idx = in_idx;
                }
            }
        }
    }

    if (max_idx != -1) {
        atomicAdd(&grad_input[max_idx], grad_output[idx]);
    }
}
#endif

template<typename T>
PoolingLayer<T>::PoolingLayer(PoolingType type, int kernel_size, int stride)
    : type_(type)
    , kernel_size_(kernel_size)
    , stride_(stride)
#ifdef ENABLE_CUDNN
    , pool_desc_(nullptr)
    , input_desc_(nullptr)
    , output_desc_(nullptr)
#endif
{
    if (stride == 0 || kernel_size == 0) {
        throw std::invalid_argument("Stride and kernel_size must be non-zero.");
    }

#ifdef ENABLE_CUDNN
    utils::CHECK_CUDNN_EX(hipdnnCreatePoolingDescriptor(&pool_desc_));
    utils::CHECK_CUDNN_EX(hipdnnCreateTensorDescriptor(&input_desc_));
    utils::CHECK_CUDNN_EX(hipdnnCreateTensorDescriptor(&output_desc_));

    hipdnnPoolingMode_t mode;
    switch (type) {
        case PoolingType::Max:
            mode = HIPDNN_POOLING_MAX;
            break;
        case PoolingType::Average:
            mode = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
            break;
        case PoolingType::MaxDeterministic:
            mode = HIPDNN_POOLING_MAX_DETERMINISTIC;
            break;
        default:
            throw std::runtime_error("Unsupported pooling type");
    }

    utils::CHECK_CUDNN_EX(hipdnnSetPooling2dDescriptor(
        pool_desc_,
        mode,
        HIPDNN_NOT_PROPAGATE_NAN,
        kernel_size, kernel_size,
        0, 0,
        stride, stride
    ));
#endif
}

template<typename T>
PoolingLayer<T>::~PoolingLayer() {
#ifdef ENABLE_CUDNN
    if (pool_desc_) utils::CHECK_CUDNN_EX(hipdnnDestroyPoolingDescriptor(pool_desc_));
    if (input_desc_) utils::CHECK_CUDNN_EX(hipdnnDestroyTensorDescriptor(input_desc_));
    if (output_desc_) utils::CHECK_CUDNN_EX(hipdnnDestroyTensorDescriptor(output_desc_));
#endif
}

template<typename T>
tensor<T> PoolingLayer<T>::forward(const tensor<T>& input) {
    int batch_size = input.shape()[0];
    int channels = input.shape()[1];
    int height = input.shape()[2];
    int width = input.shape()[3];

    if (height % stride_ != 0 || width % stride_ != 0) {
        throw std::runtime_error("PoolingLayer expects divisible dimensions");
    }

    int out_h = height / stride_;
    int out_w = width / stride_;

    tensor<T> output({ batch_size, channels, out_h, out_w });

#ifdef ENABLE_CUDNN
    // Set up input and output descriptors
    utils::CHECK_CUDNN_EX(hipdnnSetTensor4dDescriptor(input_desc_,
        HIPDNN_TENSOR_NCHW, utils::dnn_type<T>(),
        batch_size, channels, height, width));

    utils::CHECK_CUDNN_EX(hipdnnSetTensor4dDescriptor(output_desc_,
        HIPDNN_TENSOR_NCHW, utils::dnn_type<T>(),
        batch_size, channels, out_h, out_w));

    const float alpha = 1.0f, beta = 0.0f;
    utils::CHECK_CUDNN_EX(hipdnnPoolingForward(
        Cuda::current().cudnn(),
        pool_desc_,
        &alpha, input_desc_, input.data(),
        &beta, output_desc_, output.data()));
#else
    int size = output.size();
    int block = 256;
    int grid = (size + block - 1) / block;

    if (type_ == PoolingType::Average) {
        avg_pool_forward_2d<<<grid, block>>>(
            output.data(), input.data(),
            batch_size, channels, height, width,
            kernel_size_, stride_,
            out_h, out_w
        );
    } else {
        max_pool_forward_2d<<<grid, block>>>(
            output.data(), input.data(),
            batch_size, channels, height, width,
            kernel_size_, stride_,
            out_h, out_w
        );
    }

    utils::THROW_CUDA_EX();
#endif

    return output;
}

template<typename T>
tensor<T> PoolingLayer<T>::backward(const tensor<T>& grad_output, const tensor<T>& input) {
    int batch_size = input.shape()[0];
    int channels = input.shape()[1];
    int height = input.shape()[2];
    int width = input.shape()[3];

    int out_h = height / stride_;
    int out_w = width / stride_;

    tensor<T> grad_input(input.shape());
    utils::CHECK_CUDA_EX(hipMemset(grad_input.data(), 0, grad_input.size() * sizeof(T)));

#ifdef ENABLE_CUDNN
    // Set up input and output descriptors
    utils::CHECK_CUDNN_EX(hipdnnSetTensor4dDescriptor(input_desc_,
        HIPDNN_TENSOR_NCHW, utils::dnn_type<T>(),
        batch_size, channels, height, width));

    utils::CHECK_CUDNN_EX(hipdnnSetTensor4dDescriptor(output_desc_,
        HIPDNN_TENSOR_NCHW, utils::dnn_type<T>(),
        batch_size, channels, out_h, out_w));

    const float alpha = 1.0f, beta = 0.0f;
    utils::CHECK_CUDNN_EX(hipdnnPoolingBackward(
        Cuda::current().cudnn(),
        pool_desc_,
        &alpha, output_desc_, grad_output.data(),
        input_desc_, input.data(),
        &beta, input_desc_, grad_input.data()));
#else
    int size = grad_input.size();
    int block = 256;
    int grid = (size + block - 1) / block;

    if (type_ == PoolingType::Average) {
        avg_pool_backward_2d<<<grid, block>>>(
            grad_input.data(), grad_output.data(),
            batch_size, channels, height, width,
            kernel_size_, stride_,
            out_h, out_w
        );
    } else {
        max_pool_backward_2d<<<grid, block>>>(
            grad_input.data(), grad_output.data(), input.data(),
            batch_size, channels, height, width,
            kernel_size_, stride_,
            out_h, out_w
        );
    }

    utils::THROW_CUDA_EX();
#endif

    return grad_input;
}

// Explicit template instantiations
template class PoolingLayer<float>;  // FP32
// template class PoolingLayer<__half>; // FP16
// template class PoolingLayer<__hip_bfloat16>; // BF16

} // namespace dnn 