#include "hip/hip_runtime.h"
#include "dnn/layers/layer_norm_layer.cuh"
#include "dnn/utils/common.cuh"

#include <hip/hip_runtime.h>

namespace dnn {

template<typename T>
LayerNormLayer<T>::LayerNormLayer(int norm_size, float epsilon, bool affine, bool training_enabled)
    : Layer<T>(training_enabled),
      norm_size_(norm_size),
      epsilon_(epsilon),
      affine_(affine),
      gamma_({norm_size}),
      beta_({norm_size}),
      grad_gamma_({norm_size}),
      grad_beta_({norm_size})
{
    if (affine_) {
        gamma_.fill(static_cast<T>(1));
        beta_.fill(static_cast<T>(0));
    }
}

template<typename T>
__global__ void layer_norm_forward_kernel(
    const T* input, T* output,
    const T* gamma, const T* beta,
    int B, int C, float epsilon, bool affine)
{
    int b = blockIdx.x;
    int tid = threadIdx.x;

    if (b >= B || tid >= C) return;

    // Compute mean
    __shared__ float mean;
    __shared__ float var;

    if (tid == 0) {
        float sum = 0.0f;
        for (int i = 0; i < C; ++i) {
            sum += static_cast<float>(input[b * C + i]);
        }
        mean = sum / C;

        float sq_sum = 0.0f;
        for (int i = 0; i < C; ++i) {
            float val = static_cast<float>(input[b * C + i]) - mean;
            sq_sum += val * val;
        }
        var = sq_sum / C;
    }

    __syncthreads();

    float rstd = rsqrtf(var + epsilon);
    float x = static_cast<float>(input[b * C + tid]);
    float norm = (x - mean) * rstd;

    float scaled = affine ? norm * static_cast<float>(gamma[tid]) + static_cast<float>(beta[tid]) : norm;
    output[b * C + tid] = static_cast<T>(scaled);
}

template<typename T>
tensor<T> LayerNormLayer<T>::forward(const tensor<T>& input) {
    if (this->training_enabled_) {
        input_cache_ = input.clone();
    }
    const auto& shape = input.shape();
    int B = shape[0];
    int C = shape[1];
    tensor<T> output(shape);
    const int threads = norm_size_;
    layer_norm_forward_kernel<<<B, threads>>>(
        input.data(), output.data(),
        affine_ ? gamma_.data() : nullptr,
        affine_ ? beta_.data() : nullptr,
        B, C, epsilon_, affine_);
    return output;
}

template<typename T>
tensor<T> LayerNormLayer<T>::backward(const tensor<T>& grad_output) {
    if (this->training_enabled_) {
        if (!input_cache_.has_value()) {
            throw std::runtime_error("LayerNormLayer: input_cache_ is empty in backward().");
        }
    }
    return grad_output.clone();
}

template<typename T>
std::vector<tensor<T>*> LayerNormLayer<T>::parameters() {
    return affine_ ? std::vector<tensor<T>*> { &gamma_, &beta_ } : std::vector<tensor<T>*>{};
}

template<typename T>
std::vector<tensor<T>*> LayerNormLayer<T>::gradients() {
    return affine_ ? std::vector<tensor<T>*> { &grad_gamma_, &grad_beta_ } : std::vector<tensor<T>*>{};
}

template<typename T>
void LayerNormLayer<T>::save(std::ostream& out) const {
    out.write(reinterpret_cast<const char*>(&norm_size_), sizeof(norm_size_));
    gamma_.save(out);
    beta_.save(out);
    grad_gamma_.save(out);
    grad_beta_.save(out);
}

template<typename T>
void LayerNormLayer<T>::load(std::istream& in) {
    in.read(reinterpret_cast<char*>(&norm_size_), sizeof(norm_size_));
    gamma_.load(in);
    beta_.load(in);
    grad_gamma_.load(in);
    grad_beta_.load(in);
}

// Explicit instantiation
template class LayerNormLayer<float>;
template class LayerNormLayer<__half>;

}  // namespace dnn
