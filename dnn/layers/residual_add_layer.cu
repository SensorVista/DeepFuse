#include "hip/hip_runtime.h"
#include "residual_add_layer.cuh"
#include "../utils/common.cuh"

using namespace dnn::utils;

namespace dnn {

template<typename T>
ResidualAddLayer<T>::ResidualAddLayer(bool training_enabled)
    : Layer<T>(training_enabled), residual_(nullptr)
#ifdef ENABLE_CUDNN
    , desc_(nullptr)
#endif
{
#ifdef ENABLE_CUDNN
    utils::CHECK_CUDNN_EX(hipdnnCreateTensorDescriptor(&desc_));
#endif
}

template<typename T>
ResidualAddLayer<T>::~ResidualAddLayer() {
#ifdef ENABLE_CUDNN
    if (desc_) hipdnnDestroyTensorDescriptor(desc_);
#endif
}

template<typename T>
void ResidualAddLayer<T>::set_residual(tensor<T>* residual) {
    residual_ = residual;
}

template<typename T>
tensor<T> ResidualAddLayer<T>::forward(const tensor<T>& input) {
    if (this->training_enabled_) {
        input_cache_ = input.clone();
    }
    if (!residual_) {
        throw std::runtime_error("ResidualAddLayer: residual_ is not set before forward().");
    }
    if (residual_->shape() != input.shape()) {
        throw std::runtime_error("ResidualAddLayer: input and residual shapes must match.");
    }
#ifdef ENABLE_CUDNN
    tensor<T> output(input.shape());
    const int N = input.shape(0);
    const int C = input.shape(1);
    const int H = input.shape(2);
    const int W = input.shape(3);
    utils::CHECK_CUDNN_EX(hipdnnSetTensor4dDescriptor(
        desc_, HIPDNN_TENSOR_NCHW, cudnn_dtype<T>(), N, C, H, W));
    const float alpha = 1.0f;
    const float beta = 0.0f;
    utils::CHECK_CUDA_EX(hipMemcpy(output.data(), input.data(), input.size() * sizeof(T), hipMemcpyDeviceToDevice));
    const float beta_add = 1.0f;
    utils::CHECK_CUDNN_EX(hipdnnAddTensor(
        utils::cudnn_handle(), &alpha, desc_, residual_->data(), &beta_add, desc_, output.data()));
    return output;
#else
    tensor<T> output = input + *residual_;
    return output;
#endif
}

template<typename T>
tensor<T> ResidualAddLayer<T>::backward(const tensor<T>& grad_output) {
    if (this->training_enabled_) {
        if (!input_cache_.has_value()) {
            throw std::runtime_error("ResidualAddLayer: input_cache_ is empty in backward().");
        }
    }
    return grad_output.clone();
}

// Explicit template instantiations
template class ResidualAddLayer<float>;
template class ResidualAddLayer<__half>;

} // namespace dnn
