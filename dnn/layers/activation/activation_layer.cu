#include "hip/hip_runtime.h"
#include "activation_layer.cuh"
#include "../../utils/common.cuh"
#include "../../core/device.cuh"

#include <hip/hip_runtime.h>
#include <stdexcept>

#ifndef ENABLE_CUDNN
// CUDA kernels for different activation functions
template<typename T>
__global__ void relu_forward_kernel(const T* input, T* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = max(static_cast<T>(0.0f), input[idx]);
    }
}

template<typename T>
__global__ void relu_backward_kernel(const T* grad_output, const T* input, T* grad_input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_input[idx] = (input[idx] > static_cast<T>(0.0f)) ? grad_output[idx] : static_cast<T>(0.0f);
    }
}

template<typename T>
__global__ void sigmoid_kernel(const T* input, T* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = T(1.0) / (T(1.0) + exp(-input[idx]));
    }
}

template<typename T>
__global__ void sigmoid_derivative_kernel(const T* grad_output, const T* input, T* grad_input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        T sigmoid_x = T(1.0) / (T(1.0) + exp(-input[idx]));
        grad_input[idx] = grad_output[idx] * sigmoid_x * (T(1.0) - sigmoid_x);
    }
}

template<typename T>
__global__ void tanh_forward_kernel(T* output, const T* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = static_cast<T>(1.7159f * tanh(2.0f/3.0f * input[idx]));
    }
}

template<typename T>
__global__ void tanh_backward_kernel(T* grad_input, const T* grad_output, const T* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        T tanh_sx = tanh(2.0f/3.0f * input[idx]);
        T activation_derivative = static_cast<T>(1.7159f * (2.0f/3.0f) * (1.0f - tanh_sx * tanh_sx));
        grad_input[idx] = grad_output[idx] * activation_derivative;
    }
}
#endif

namespace dnn {

template<typename T>
ActivationLayer<T>::ActivationLayer(ActivationType type) : type_(type) {
#ifdef ENABLE_CUDNN
    utils::CHECK_CUDNN_EX(hipdnnCreateActivationDescriptor(&act_desc_));
    hipdnnActivationMode_t mode;
    switch (type) {
        case ActivationType::ReLU:
            mode = HIPDNN_ACTIVATION_RELU;
            break;
        case ActivationType::Sigmoid:
            mode = HIPDNN_ACTIVATION_SIGMOID;
            break;
        case ActivationType::Tanh:
            mode = HIPDNN_ACTIVATION_TANH;
            break;
        case ActivationType::ClippedReLU:
            mode = HIPDNN_ACTIVATION_CLIPPED_RELU;
            break;
        case ActivationType::Elu:
            mode = HIPDNN_ACTIVATION_ELU;
            break;
        default:
            throw std::runtime_error("Unsupported activation type");
    }
    utils::CHECK_CUDNN_EX(hipdnnSetActivationDescriptor(act_desc_, mode, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
#endif
}

// Helper constexpr mapping
constexpr const char* activation_type_to_string(ActivationType type) {
    switch (type) {
        case ActivationType::ReLU: return "ReLU";
        case ActivationType::Sigmoid: return "Sigmoid";
        case ActivationType::Tanh: return "Tanh";
        case ActivationType::ClippedReLU: return "ClippedReLU";
        case ActivationType::Elu: return "Elu";
        default: return "Unknown";
    }
}

template<typename T>
std::string ActivationLayer<T>::name() const { return "Activation(" + std::string(activation_type_to_string(type_)) + ")"; }

template<typename T>
ActivationLayer<T>::~ActivationLayer() {
#ifdef ENABLE_CUDNN
    utils::CHECK_CUDNN_EX(hipdnnDestroyActivationDescriptor(act_desc_));
#endif
}

template<typename T>
tensor<T> ActivationLayer<T>::forward(const tensor<T>& input) {
    tensor<T> output(input.shape());
    int size = input.size();
    
    if (size == 0) return output;

#ifdef ENABLE_CUDNN
    const float alpha = 1.0f, beta = 0.0f;
    auto handle = Cuda::current().cudnn();
    utils::CHECK_CUDNN_EX(hipdnnActivationForward(handle, act_desc_, &alpha, 
        input.desc(), input.data(), &beta, output.desc(), output.data()));
#else
    const int BLOCK_SIZE = 256;
    int num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    switch (type_) {
        case ActivationType::ReLU:
            relu_forward_kernel<<<num_blocks, BLOCK_SIZE>>>(input.data(), output.data(), size);
            break;
        case ActivationType::Sigmoid:
            sigmoid_kernel<<<num_blocks, BLOCK_SIZE>>>(input.data(), output.data(), size);
            break;
        case ActivationType::Tanh:
            tanh_forward_kernel<<<num_blocks, BLOCK_SIZE>>>(output.data(), input.data(), size);
            break;
        default:
            throw std::runtime_error("Unsupported activation type");
    }
    utils::THROW_CUDA_EX();
#endif

    return output;
}

template<typename T>
tensor<T> ActivationLayer<T>::backward(const tensor<T>& grad_output, const tensor<T>& input) {
    tensor<T> grad_input(grad_output.shape());
    int size = grad_output.size();

    if (size == 0) return grad_input;

#ifdef ENABLE_CUDNN
    const float alpha = 1.0f, beta = 0.0f;
    auto handle = Cuda::current().cudnn();
    utils::CHECK_CUDNN_EX(hipdnnActivationBackward(handle, act_desc_, &alpha,
        grad_output.desc(), grad_output.data(), grad_output.desc(), grad_output.data(),
        input.desc(), input.data(), &beta, grad_input.desc(), grad_input.data()));
#else
    const int BLOCK_SIZE = 256;
    int num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    switch (type_) {
        case ActivationType::ReLU:
            relu_backward_kernel<<<num_blocks, BLOCK_SIZE>>>(grad_output.data(), input.data(), grad_input.data(), size);
            break;
        case ActivationType::Sigmoid:
            sigmoid_derivative_kernel<<<num_blocks, BLOCK_SIZE>>>(grad_output.data(), input.data(), grad_input.data(), size);
            break;
        case ActivationType::Tanh:
            tanh_backward_kernel<<<num_blocks, BLOCK_SIZE>>>(grad_input.data(), grad_output.data(), input.data(), size);
            break;
        default:
            throw std::runtime_error("Unsupported activation type");
    }
    utils::THROW_CUDA_EX();
#endif

    return grad_input;
}

} // namespace dnn 